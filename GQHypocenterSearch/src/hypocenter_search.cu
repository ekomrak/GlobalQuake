#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <jni.h>
#include <stdio.h>

#include "globalquake.hpp"
#include "globalquake_jni_GQNativeFunctions.h"
#include "travel_table.hpp"

#define BLOCK_REDUCE 256
#define BLOCK_DISTANCES 64
#define SHARED_TRAVEL_TABLE_SIZE 256

#define STATION_FILEDS 4
#define HYPOCENTER_FILEDS 4
/**
 * STATION:
 * lat | lon | alt | pwave
 * 
 * PRELIMINARY_HYPOCENTER (STEP 1):
 * heuristic | position | origin
 * 
 * PRELIMINARY_HYPOCENTER (STEP 2):
 * heuristic | index (int) | depth | origin
 * 
 * RESULT_HYPOCENTER:
 * lat, lon, depth, origin
*/

#define MAX_ANG_VIRTUAL (181.0f)
#define PHI2 2.618033989f
#define PI 3.14159256f

struct depth_profile_t
{
    float depth_resolution;
    float *device_travel_table;
};

bool cuda_initialised = false;
float max_depth_resolution;

int depth_profile_count;
depth_profile_t *depth_profiles = nullptr;
float *f_results_device = nullptr;

size_t total_travel_table_size;

void print_err(const char *msg) {
    hipError_t err = hipGetLastError();
    TRACE(2, "%s failed: %s (%d)\n", msg, hipGetErrorString(err), err);
}

__host__ void move_on_globe(float from_lat, float from_lon, float angle, float angular_distance, float *lat, float *lon) {
    // calculate angles
    float delta = angular_distance;
    float theta = from_lat;
    float phi = from_lon;
    float gamma = angle;

    // calculate sines and cosines
    float c_theta = cosf(theta);
    float s_theta = sinf(theta);
    float c_phi = cosf(phi);
    float s_phi = sinf(phi);
    float c_delta = cosf(delta);
    float s_delta = sinf(delta);
    float c_gamma = cosf(gamma);
    float s_gamma = sinf(gamma);

    // calculate end vector
    float x = c_delta * c_theta * c_phi - s_delta * (s_theta * c_phi * c_gamma + s_phi * s_gamma);
    float y = c_delta * c_theta * s_phi - s_delta * (s_theta * s_phi * c_gamma - c_phi * s_gamma);
    float z = s_delta * c_theta * c_gamma + c_delta * s_theta;

    // calculate end lat long
    *lat = asinf(z);
    *lon = atan2f(y, x);
}

__device__ void move_on_globe_device(float from_lat, float from_lon, float angle, float angular_distance, float *lat, float *lon) {
    // calculate angles
    float delta = angular_distance;
    float theta = from_lat;
    float phi = from_lon;
    float gamma = angle;

    // calculate sines and cosines
    float c_theta = __cosf(theta);
    float s_theta = __sinf(theta);
    float c_phi = __cosf(phi);
    float s_phi = __sinf(phi);
    float c_delta = __cosf(delta);
    float s_delta = __sinf(delta);
    float c_gamma = __cosf(gamma);
    float s_gamma = __sinf(gamma);

    // calculate end vector
    float x = c_delta * c_theta * c_phi - s_delta * (s_theta * c_phi * c_gamma + s_phi * s_gamma);
    float y = c_delta * c_theta * s_phi - s_delta * (s_theta * s_phi * c_gamma - c_phi * s_gamma);
    float z = s_delta * c_theta * c_gamma + c_delta * s_theta;

    // calculate end lat long
    *lat = asinf(z);
    *lon = atan2f(y, x);
}

// everything is in radians
__device__ float haversine(float lat1, float lon1, float lat2, float lon2) {
    float dlat = lat2 - lat1;
    float dlon = lon2 - lon1;

    // Haversine formula
    float v1 = __sinf(dlat / 2.0f);
    float v2 = __sinf(dlon / 2.0f);
    float a = v1 * v1 + __cosf(lat1) * __cosf(lat2) * v2 * v2;

    float y = __fsqrt_rn(a);
    float x = __fsqrt_rn(1.0f - a);
    float c = atan2f(y, x); // bottleneck of station distances calculation

    return c * 2.0f; // Angular distance in radians
}

// everything in radians
void calculate_params(int points, int index, float max_dist, float from_lat, float from_lon, float *lat, float *lon, float *dist) {
    float ang = (2.0f * PI * (float) index) / PHI2;
    *dist = sqrtf(index) * (max_dist / sqrtf(points - 1.0f));
    move_on_globe(from_lat, from_lon, ang, *dist, lat, lon);
}

__device__ void calculate_params_device(int points, int index, float max_dist, float from_lat, float from_lon, float *lat, float *lon, float *dist) {
    float ang = (2.0f * PI * (float) index) / PHI2;
    *dist = __fsqrt_rn(index) * (max_dist / __fsqrt_rn(points - 1.0f));
    move_on_globe_device(from_lat, from_lon, ang, *dist, lat, lon);
}

__device__ float travel_table_interpolate(float *s_travel_table, float index) {
    int index1 = (int) index;
    int index2 = index1 + 1;

    float t = index - index1;
    return (1.0f - t) * s_travel_table[index1] + t * s_travel_table[index2];
}

__device__ inline float *hypocenter_heuristic(float *hypocenter, int grid_size) {
    return &hypocenter[0 * grid_size];
}

__device__ inline float *hypocenter_index(float *hypocenter, int grid_size) {
    return &hypocenter[1 * grid_size];
}

__device__ inline float *hypocenter_depth_index(float *hypocenter, int grid_size) {
    return &hypocenter[2 * grid_size];
}

__device__ inline float *hypocenter_origin(float *hypocenter, int grid_size) {
    return &hypocenter[3 * grid_size];
}

__device__ inline float heuristic(float correct, float err) {
    return (correct * correct) / (err * err);
}

__device__ void reduce(float *hypocenter_a, float *hypocenter_b, int grid_size) {
    float heuristic_a = *hypocenter_heuristic(hypocenter_a, grid_size);
    float heuristic_b = *hypocenter_heuristic(hypocenter_b, grid_size);

    bool swap = heuristic_b > heuristic_a;

    if (swap) {
        *hypocenter_heuristic(hypocenter_a, grid_size) = *hypocenter_heuristic(hypocenter_b, grid_size);
        *hypocenter_depth_index(hypocenter_a, grid_size) = *hypocenter_depth_index(hypocenter_b, grid_size);
        *hypocenter_index(hypocenter_a, grid_size) = *hypocenter_index(hypocenter_b, grid_size);
        *hypocenter_origin(hypocenter_a, grid_size) = *hypocenter_origin(hypocenter_b, grid_size);
    }
}

__global__ void evaluate_hypocenter(float *results,
        float *travel_table,
        float *stations,
        float *station_distances,
        float *station_distances_across,
        int station_count,
        int points,
        float max_dist,
        float p_wave_threshold) {
    extern __shared__ float s_stations[];
    __shared__ float s_travel_table[SHARED_TRAVEL_TABLE_SIZE * TILE];
    __shared__ float s_results[BLOCK_HYPOCS * HYPOCENTER_FILEDS];

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;


    for (int station_iteration = 0; station_iteration < ceilf(static_cast<float>(station_count * 1) / blockDim.x); station_iteration++) {
        int index = station_iteration * blockDim.x + threadIdx.x;

        if (index < station_count * 1) {
            s_stations[index] = stations[index + 3 * station_count]; // we care only P wave
        }
    }

    for(int tile = 0; tile < TILE; tile++) {
        for (int tt_iteration = 0; tt_iteration < ceilf((SHARED_TRAVEL_TABLE_SIZE) / static_cast<float>(blockDim.x)); tt_iteration++) {
            int s_index = tt_iteration * blockDim.x + threadIdx.x;
            if (s_index < SHARED_TRAVEL_TABLE_SIZE) {
                s_travel_table[tile *SHARED_TRAVEL_TABLE_SIZE + s_index] = travel_table[(blockIdx.y * TILE + tile) * SHARED_TRAVEL_TABLE_SIZE + s_index];
            }
        }
    }

    __syncthreads();

    float origins[TILE];

    int j = (point_index) % station_count;

    // trick with changing station that is being used for origin calculation
    {
        float ang_dist = station_distances_across[point_index];
        float s_pwave = s_stations[j];

        for(int tile = 0; tile < TILE; tile++) {
            float expected_travel_time = travel_table_interpolate(&s_travel_table[tile * SHARED_TRAVEL_TABLE_SIZE], ang_dist);
            float predicted_origin = s_pwave - expected_travel_time;

            origins[tile] = predicted_origin;
        }
    }
    
    float err[TILE];
    float correct[TILE];
    for(int tile = 0; tile < TILE; tile++) {
        err[tile] = 0.0f;
        correct[tile] = 0.0f;
    }

    for (int i = 0; i < station_count; i++) {
        float ang_dist = station_distances[point_index + i * points];
        float s_pwave = s_stations[i];

        for(int tile = 0; tile < TILE; tile++) {
            float expected_travel_time = travel_table_interpolate(&s_travel_table[tile * SHARED_TRAVEL_TABLE_SIZE], ang_dist);
            float predicted_origin = s_pwave - expected_travel_time;

            float _err = fabsf(predicted_origin - origins[tile]);
            correct[tile] += fmaxf(0.0f, p_wave_threshold - _err); // divide by p_wave_threshold at the end! ! actually we dont have to
            err[tile] += _err;
        }
    }

    int best_tile = 0;
    float best_heuristic = heuristic(correct[0], err[0]);
    
    #if TILE > 1
    for(int tile = 1; tile < TILE; tile++) {
        float h = heuristic(correct[tile], err[tile]);
        if(h > best_heuristic){
            best_heuristic = h;
            best_tile = tile;
        }
    }
    #endif
    
    float depth = blockIdx.y * TILE + best_tile;
            
    s_results[threadIdx.x + blockDim.x * 0] = best_heuristic;
    *(int *) (&s_results[threadIdx.x + blockDim.x * 1]) = point_index;
    s_results[threadIdx.x + blockDim.x * 2] = depth;
    s_results[threadIdx.x + blockDim.x * 3] = origins[best_tile];

    __syncthreads();

    // implementation 3 from slides
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && blockDim.x * blockIdx.x + threadIdx.x + s < points) {
            reduce(&s_results[threadIdx.x], &s_results[threadIdx.x + s], blockDim.x);
            __syncthreads();
        }
    }

    if (threadIdx.x == 0) {
        int idx = (blockIdx.y) * gridDim.x + blockIdx.x;

        results[idx + 0 * (gridDim.x * gridDim.y)] = s_results[0 * blockDim.x]; // heuristic
        results[idx + 1 * (gridDim.x * gridDim.y)] = s_results[1 * blockDim.x]; // point_index
        results[idx + 2 * (gridDim.x * gridDim.y)] = s_results[2 * blockDim.x]; // depth
        results[idx + 3 * (gridDim.x * gridDim.y)] = s_results[3 * blockDim.x]; // origin
    }
}

__global__ void results_reduce(float *out, float *in, int total_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= total_size) {
        return;
    }
    __shared__ float s_results[HYPOCENTER_FILEDS * BLOCK_REDUCE];

    s_results[threadIdx.x + BLOCK_REDUCE * 0] = in[index + total_size * 0];
    s_results[threadIdx.x + BLOCK_REDUCE * 1] = in[index + total_size * 1];
    s_results[threadIdx.x + BLOCK_REDUCE * 2] = in[index + total_size * 2];
    s_results[threadIdx.x + BLOCK_REDUCE * 3] = in[index + total_size * 3];
    __syncthreads();

    // implementation 3 from slides
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && blockDim.x * blockIdx.x + threadIdx.x + s < total_size) {
            reduce(&s_results[threadIdx.x], &s_results[threadIdx.x + s], blockDim.x);
            __syncthreads();
        }
    }

    if (threadIdx.x == 0) {
        int idx = blockIdx.y * gridDim.x + blockIdx.x;
        out[idx + 0 * (gridDim.x * gridDim.y)] = s_results[0 * blockDim.x];
        out[idx + 1 * (gridDim.x * gridDim.y)] = s_results[1 * blockDim.x];
        out[idx + 2 * (gridDim.x * gridDim.y)] = s_results[2 * blockDim.x];
        out[idx + 3 * (gridDim.x * gridDim.y)] = s_results[3 * blockDim.x];
    }
}

const float ANGLE_TO_INDEX = (SHARED_TRAVEL_TABLE_SIZE - 1.0f) / MAX_ANG_VIRTUAL;

__global__ void precompute_station_distances(
        float *station_distances, float* station_distances_across, float *stations, int station_count, int points, float max_dist, float from_lat, float from_lon) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= points) {
        return;
    }

    float lat, lon, dist;

    calculate_params_device(points, index, max_dist, from_lat, from_lon, &lat, &lon, &dist);

    int j = index % station_count;

    for (int i = 0; i < station_count; i++) {        
        float s_lat = stations[i + 0 * station_count];
        float s_lon = stations[i + 1 * station_count];
        float ang_dist = haversine(lat, lon, s_lat, s_lon) * 180.0f / PI;  // because travel table is in degrees
        float ang_index = ang_dist * ANGLE_TO_INDEX; // precompute;
        station_distances[index + i * points] = ang_index;

        if( i == j ) {
            station_distances_across[index] = ang_index;
        }
    }
}

void prepare_travel_table(float *fitted_travel_table, int rows) {
    for (int row = 0; row < rows; row++) {
        for (int column = 0; column < SHARED_TRAVEL_TABLE_SIZE; column++) {
            fitted_travel_table[row * SHARED_TRAVEL_TABLE_SIZE + column] =
                    p_wave_interpolate(column / (SHARED_TRAVEL_TABLE_SIZE - 1.0) * MAX_ANG_VIRTUAL, (row / (rows - 1.0)) * table_max_depth);
        }
    }
}

// returns (accurately) estimated total GPU memory allocation size given the parameters
size_t get_total_allocation_size(size_t points, size_t station_count, float depth_resolution) {
    size_t result = total_travel_table_size;

    dim3 blocks = { (unsigned int) ceil(static_cast<float>(points) / BLOCK_HYPOCS), (unsigned int) ceil(table_max_depth / (depth_resolution * TILE)) + 1, 1 };

    size_t station_array_size = sizeof(float) * station_count * STATION_FILEDS;
    size_t station_distances_array_size = sizeof(float) * station_count * points;
    size_t results_size = sizeof(float) * HYPOCENTER_FILEDS * (blocks.x * blocks.y * blocks.z);

    size_t temp_results_array_elements = ceil((blocks.x * blocks.y * blocks.z) / static_cast<float>(BLOCK_REDUCE));
    size_t temp_results_array_size = (sizeof(float) * HYPOCENTER_FILEDS * temp_results_array_elements);

    result += station_array_size;
    result += station_distances_array_size;
    result += results_size;
    result += temp_results_array_size;

    return result;
}

JNIEXPORT jlong JNICALL Java_globalquake_jni_GQNativeFunctions_getAllocationSize(JNIEnv *, jclass, jint points, jint stations, jfloat depth_resolution) {
    return get_total_allocation_size(points, stations, depth_resolution);
}

bool run_hypocenter_search(float *stations,
        size_t station_count,
        size_t points,
        int depth_profile_index,
        float max_dist,
        float from_lat,
        float from_lon,
        float *final_result,
        float p_wave_threshold) {
    if (depth_profile_index < 0 || depth_profile_index >= depth_profile_count) {
        TRACE(2, "Error! Invalid depth profile index: %d!\n", depth_profile_index);
        return false;
    }

    depth_profile_t *depth_profile = &depth_profiles[depth_profile_index];

    float *device_stations;
    float *device_stations_distances;
    float *device_stations_distances_across;
    float *device_temp_results;

    if (points < 2) {
        TRACE(2, "Error! at least 2 points needed!\n");
        return false;
    }

    if (station_count < 3) {
        TRACE(2, "Error! at least 3 stations needed!\n");
        return false;
    }

    points += (BLOCK_HYPOCS - points % BLOCK_HYPOCS);

    bool success = true;

    dim3 blocks = {
        (unsigned int) ceil(static_cast<float>(points) / BLOCK_HYPOCS), (unsigned int) ceil(table_max_depth / (depth_profile->depth_resolution * TILE)) + 1, 1
    };
    dim3 threads = { BLOCK_HYPOCS, 1, 1 };

    if (blocks.y < 2) {
        TRACE(2, "Error! at least 2 depth points needed!\n");
        return false;
    }

    size_t station_array_size = sizeof(float) * station_count * STATION_FILEDS;
    size_t station_distances_array_size = sizeof(float) * station_count * points;
    size_t station_distances_array_size_across = sizeof(float) * points;
    size_t results_size = sizeof(float) * HYPOCENTER_FILEDS * (blocks.x * (blocks.y) * blocks.z);

    size_t temp_results_array_elements = ceil((blocks.x * (blocks.y ) * blocks.z) / static_cast<float>(BLOCK_REDUCE));
    size_t current_result_count = blocks.x * (blocks.y) * blocks.z;

    const int block_count = ceil(static_cast<float>(points) / BLOCK_DISTANCES);

    TRACE(1, "Station array size (%ld stations) %.2fkB\n", station_count, station_array_size / (1024.0));
    TRACE(1, "Station distances array size %.2fMB\n", station_distances_array_size / (1024.0 * 1024.0));
    TRACE(1, "Temp results array size %.2fkB\n", (sizeof(float) * HYPOCENTER_FILEDS * temp_results_array_elements) / (1024.0));
    TRACE(1, "Results array has size %.2fMB\n", (results_size / (1024.0 * 1024.0)));

    success &= hipMalloc(&device_stations, station_array_size) == hipSuccess;
    success &= hipMemcpy(device_stations, stations, station_array_size, hipMemcpyHostToDevice) == hipSuccess;
    success &= hipMalloc(&device_stations_distances, station_distances_array_size) == hipSuccess;
    success &= hipMalloc(&device_stations_distances_across, station_distances_array_size_across) == hipSuccess;
    success &= hipMalloc(&device_temp_results, sizeof(float) * HYPOCENTER_FILEDS * temp_results_array_elements) == hipSuccess;
    success &= hipMalloc(&f_results_device, results_size) == hipSuccess;

    if (!success) {
        print_err("Hypocenter search initialisation");
        goto cleanup;
    }

    TRACE(1, "Grid size: %d %d %d\n", blocks.x, blocks.y, blocks.z);
    TRACE(1, "Block size: %d %d %d\n", threads.x, threads.y, threads.z);
    TRACE(1, "Total points: %lld\n", (((long long) (blocks.x * blocks.y * blocks.z)) * (long long) (threads.x * threads.y * threads.z)));

    if (success) {
        precompute_station_distances<<<block_count, BLOCK_DISTANCES>>>(
                device_stations_distances, device_stations_distances_across, device_stations, station_count, points, max_dist, from_lat, from_lon);
    }

    success &= hipDeviceSynchronize() == hipSuccess;

    if (!success) {
        print_err("Calculate station distances");
        goto cleanup;
    }

    if (success) {
        evaluate_hypocenter<<<blocks, threads, sizeof(float) * station_count>>>(f_results_device,
                depth_profile->device_travel_table,
                device_stations,
                device_stations_distances,
                device_stations_distances_across,
                station_count,
                points,
                max_dist,
                p_wave_threshold);
    }

    success &= hipDeviceSynchronize() == hipSuccess;

    if (!success) {
        print_err("Hypocenter search");
        goto cleanup;
    }

    while (success && current_result_count > 1) {
        dim3 blocks_reduce = { (unsigned int) ceil(current_result_count / static_cast<double>(BLOCK_REDUCE)), 1, 1 };
        TRACE(1, "Reducing... from %ld to %d\n", current_result_count, blocks_reduce.x);

        results_reduce<<<blocks_reduce, BLOCK_REDUCE>>>(device_temp_results, f_results_device, current_result_count);
        success &= hipDeviceSynchronize() == hipSuccess;

        if (!success) {
            print_err("Reduce");
            goto cleanup;
        }

        current_result_count = blocks_reduce.x;

        float local_result[HYPOCENTER_FILEDS];

        if (current_result_count == 1) {
            success &= hipMemcpy(local_result, device_temp_results, HYPOCENTER_FILEDS * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess;

            float lat, lon, u_dist;
            calculate_params(points, *(int *) &local_result[1], max_dist, from_lat, from_lon, &lat, &lon, &u_dist);

            double depth = table_max_depth * (local_result[2] / (float) (blocks.y * TILE - 1.0f));

            final_result[0] = lat; // lat
            final_result[1] = lon; // lon
            final_result[2] = depth;
            final_result[3] = local_result[3]; // origin
        } else {
            success &= hipMemcpy(f_results_device, device_temp_results, current_result_count * HYPOCENTER_FILEDS * sizeof(float), hipMemcpyDeviceToDevice) ==
                    hipSuccess;
        }

        if (!success) {
            print_err("CUDA memcpy");
            goto cleanup;
        }
    }

cleanup:

    if (device_stations) {
        success &= hipFree(device_stations) == hipSuccess;
    }
    if (device_stations_distances_across) {
        success &= hipFree(device_stations_distances_across) == hipSuccess;
    }
    if (device_stations_distances) {
        success &= hipFree(device_stations_distances) == hipSuccess;
    }
    if (device_temp_results) {
        success &= hipFree(device_temp_results) == hipSuccess;
    }
    if (f_results_device) {
        success &= hipFree(f_results_device) == hipSuccess;
    }

    return success;
}

JNIEXPORT jfloatArray JNICALL Java_globalquake_jni_GQNativeFunctions_findHypocenter(JNIEnv *env,
        jclass,
        jfloatArray stations,
        jfloat from_lat,
        jfloat from_lon,
        jlong points,
        int depth_resolution_profile_id,
        jfloat max_dist,
        jfloat p_wave_threshold) {
    size_t station_count = env->GetArrayLength(stations) / STATION_FILEDS;

    float *stations_array = static_cast<float *>(malloc(sizeof(float) * station_count * STATION_FILEDS));
    if (!stations_array) {
        perror("malloc");
        return nullptr;
    }

    jfloat *elements = env->GetFloatArrayElements(stations, 0);
    for (int i = 0; i < station_count * STATION_FILEDS; i++) {
        stations_array[i] = elements[i];
    }

    env->ReleaseFloatArrayElements(stations, elements, 0);

    float final_result[HYPOCENTER_FILEDS];

    bool success = run_hypocenter_search(
            stations_array, station_count, points, depth_resolution_profile_id, max_dist, from_lat, from_lon, final_result, p_wave_threshold);

    free(stations_array);

    jfloatArray result = nullptr;

    if (success) {
        result = env->NewFloatArray(4);

        if (result != nullptr) {
            env->SetFloatArrayRegion(result, 0, 4, final_result);
        }
    }

    return result;
}

bool init_depth_profiles(float *resols, int count) {
    max_depth_resolution = table_max_depth;
    depth_profile_count = count;

    depth_profiles = static_cast<depth_profile_t *>(malloc(count * sizeof(depth_profile_t)));
    if (depth_profiles == nullptr) {
        perror("malloc");
        return false;
    }

    total_travel_table_size = 0;

    for (int i = 0; i < depth_profile_count; i++) {
        float depth_resolution = resols[i];
        if (depth_resolution < max_depth_resolution) {
            max_depth_resolution = depth_resolution;
        }

        depth_profiles[i].depth_resolution = depth_resolution;

        int rows = (unsigned int) ceil(table_max_depth / depth_resolution) + 1;
        size_t table_size = sizeof(float) * rows * SHARED_TRAVEL_TABLE_SIZE;
        total_travel_table_size += table_size;

        TRACE(1, "Creating depth profile with resolution %.2fkm (%.2fkB)\n", depth_resolution, table_size / 1024.0);

        // todo fitted array
        if (hipMalloc(&depth_profiles[i].device_travel_table, table_size) != hipSuccess) {
            print_err("CUDA malloc");
            return false;
        }

        float *fitted_travel_table = static_cast<float *>(malloc(table_size));

        if (fitted_travel_table == nullptr) {
            perror("malloc");
            return false;
        } else {
            prepare_travel_table(fitted_travel_table, rows);
            if (hipMemcpy(depth_profiles[i].device_travel_table, fitted_travel_table, table_size, hipMemcpyHostToDevice) != hipSuccess) {
                print_err("CUDA memcpy");
                free(fitted_travel_table);
                return false;
            }

            free(fitted_travel_table);
            fitted_travel_table = NULL;
        }
    }

    return true;
}

/*
 * Class:     globalquake_jni_GQNativeFunctions
 * Method:    initCUDA
 * Signature: ()Z
 */
JNIEXPORT jboolean JNICALL Java_globalquake_jni_GQNativeFunctions_initCUDA(JNIEnv *env, jclass, jfloatArray depth_profiles_array) {
    bool success = true;

    if (depth_profiles_array != nullptr && depth_profiles == nullptr) {
        int depth_profile_count = env->GetArrayLength(depth_profiles_array);
        jfloat *depth_resolutions_array = env->GetFloatArrayElements(depth_profiles_array, 0);

        float depth_resolutions[depth_profile_count];
        for (int i = 0; i < depth_profile_count; i++) {
            depth_resolutions[i] = depth_resolutions_array[i];
        }

        env->ReleaseFloatArrayElements(depth_profiles_array, depth_resolutions_array, 0);

        success &= init_depth_profiles(depth_resolutions, depth_profile_count);
    }

    cuda_initialised = success;
    return success;
}
